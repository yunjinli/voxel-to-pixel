#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void backward_kernel(
    const float *grad_output, const int *depth_buffer, const float *features,
    const float *coordinates, float *grad_features, const float *projection_matrix,
    int N, int C, int H, int W)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    // Load coordinates
    float z = coordinates[idx * 3];
    float y = coordinates[idx * 3 + 1];
    float x = coordinates[idx * 3 + 2];

    // Apply the projection matrix (assume saved in shared memory for performance)
    // extern __shared__ float projection_matrix[];
    float u = projection_matrix[0] * x + projection_matrix[1] * y + projection_matrix[2] * z + projection_matrix[3];
    float v = projection_matrix[4] * x + projection_matrix[5] * y + projection_matrix[6] * z + projection_matrix[7];
    float depth = projection_matrix[8] * x + projection_matrix[9] * y + projection_matrix[10] * z + projection_matrix[11];

    // Map to pixel coordinates
    int u_d = int(u / depth);
    int v_d = int(v / depth);
    if (u_d < 0 || u_d > W)
        return;
    if (v_d < 0 || v_d > H)
        return;
    if (depth < 0)
        return;
    int pixel_idx = v_d * W + u_d;
    // Check depth buffer for the closest voxel at this pixel
    if (__int_as_float(depth_buffer[pixel_idx]) != depth)
        return;

    // Backpropagate feature gradients
    for (int c = 0; c < C; ++c)
    {
        atomicAdd(&grad_features[idx * C + c], grad_output[pixel_idx * C + c]);
    }
}

torch::Tensor backward(
    torch::Tensor grad_output, torch::Tensor features, torch::Tensor coordinates,
    torch::Tensor depth_buffer, torch::Tensor projection_matrix, std::vector<int64_t> output_size)
{

    int N = features.size(0);
    int C = features.size(1);
    int H = output_size[0];
    int W = output_size[1];

    auto grad_features = torch::zeros_like(features);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    backward_kernel<<<blocks, threads, sizeof(float) * 12>>>(
        grad_output.data_ptr<float>(), depth_buffer.data_ptr<int>(), features.data_ptr<float>(),
        coordinates.data_ptr<float>(), grad_features.data_ptr<float>(), projection_matrix.data_ptr<float>(),
        N, C, H, W);

    return grad_features;
}